#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_plan.h"


#define CUFFTTYPE(type) ( (type) == 0 ? HIPFFT_R2C : \
                          (type) == 1 ? HIPFFT_C2R : \
                          (type) == 2 ? HIPFFT_C2C : \
                          (type) == 3 ? HIPFFT_D2Z : \
                          (type) == 4 ? HIPFFT_Z2D : \
                          (type) == 5 ? HIPFFT_Z2Z : HIPFFT_Z2Z )


hipfftHandle *cufft_plan(planlen extent,
                        int fft_type,
                        int batch_size)
{
	  hipfftHandle *plan = (hipfftHandle *)malloc(sizeof(hipfftHandle));

    int ndims = 0;
    (extent[0] > 1) ? ndims += 1 : false;
    (extent[1] > 1) ? ndims += 1 : false;
    (extent[2] > 1) ? ndims += 1 : false;

    int *dims = (int *)malloc(sizeof(int)*ndims);
    for (int i = 0; i < ndims; ++i) {
        dims[i] = extent[ndims-i-1];
    }

    gpuFFTErrchk(hipfftPlanMany(plan, ndims, dims, NULL, 0, 0, NULL, 0, 0, CUFFTTYPE(fft_type), batch_size));
    hipDeviceSynchronize();
    free(dims);

    return plan;
}
