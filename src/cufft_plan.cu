#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_plan.h"


#define CUFFTTYPE(type) ( (type) == 0 ? HIPFFT_R2C : \
                          (type) == 1 ? HIPFFT_C2R : \
                          (type) == 2 ? HIPFFT_C2C : \
                          (type) == 3 ? HIPFFT_D2Z : \
                          (type) == 4 ? HIPFFT_Z2D : \
                          (type) == 5 ? HIPFFT_Z2Z : HIPFFT_Z2Z )


hipfftHandle *cufft_plan(planlen extent,
                        int fft_type,
                        int batch_size)
{
	hipfftHandle *plan = new hipfftHandle;

    int ndims = 0;
    (extent[0] > 1) ? ndims += 1 : false;
    (extent[1] > 1) ? ndims += 1 : false;
    (extent[2] > 1) ? ndims += 1 : false;

    int *dims = new int[ndims];
    for (int i = 0; i < ndims; ++i) {
        dims[i] = extent[ndims-i-1];
    }

    gpuFFTErrchk(hipfftPlanMany(plan, ndims, dims, NULL, 0, 0, NULL, 0, 0, CUFFTTYPE(fft_type), batch_size));
    hipDeviceSynchronize();
    delete[] dims;

    return plan;
}


void cufft_plan_destroy(hipfftHandle *plan)
{
    gpuFFTErrchk(hipfftDestroy(*plan));
    delete[] plan;
}