#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_addredundants.h"

/* This will generate an array that adds the redundant values of a
CUDA R2C transformation. This function was found here:
https://devtalk.nvidia.com/default/topic/488433/cufft-only-gives-non-redundant-results/
*/
template<typename T>
__global__ void k_makeRedundant(T* dst, const T* src, int w, int h)
{
    volatile int gid_x = threadIdx.x + blockIdx.x * blockDim.x;
    volatile int gid_y = threadIdx.y + blockIdx.y * blockDim.y;
    volatile int nbNoRedundants = (w >> 1) + 1;

    // index for reading :
    volatile int gid = gid_x + nbNoRedundants * gid_y;
    T val;

    if(gid_x < nbNoRedundants && gid_y < h) {
        // write the non redundant part in the new array :
        val = src[gid];
        gid = gid_x + w * gid_y; // new index for writing
        dst[gid] = val;
    }

    // shift'n'flip
    gid_x = w - gid_x;

    if(gid_y != 0) {
        gid_y = h - gid_y;
    }

    gid = gid_x + w * gid_y;

    // write conjugate :

    if(gid_x >= nbNoRedundants && gid_x < w && gid_y >= 0 && gid_y < h) {
        val.y = -val.y;
        dst[gid] = val; // never coalesced with compute <= 1.1 ; coalesced if >= 1.2 AND w multiple of 16 AND good call configuration
    }
}


/* C compatible version that requires a dtype_id to be converted
to the proper data type. */
void cufft_addredundants(const void *d_idata,
                         void *d_odata,
                         int nx, int ny,
                         int dtype,
                         hipStream_t *stream)
{
    dim3 blockSize(16,16);
    dim3 gridSize((nx-1)/blockSize.x+1,
                  (ny-1)/blockSize.y+1);

    hipStream_t stream_id;
    (stream == NULL) ? stream_id = NULL : stream_id = *stream;

    switch(dtype) {

        case 2: {
            k_makeRedundant<<<gridSize, blockSize, 0, stream_id>>>((float2*)d_odata,
                                                                   (float2*)d_idata,
                                                                   nx, ny);
            break;
        }

        case 3: {
            k_makeRedundant<<<gridSize, blockSize, 0, stream_id>>>((double2*)d_odata,
                                                                   (double2*)d_idata,
                                                                   nx, ny);
            break;
        }
    }

    return;
}
