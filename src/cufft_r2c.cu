#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_r2c.h"


void cufft_r2c(hipfftHandle *plan,
               void *d_idata,
               void *d_odata,
               int dtype)
{
    switch(dtype) {

        case 2:
            gpuFFTErrchk(hipfftExecR2C(*plan,
                                      static_cast<float*>(d_idata),
                                      static_cast<float2*>(d_odata)));
            break;

        case 3:
            gpuFFTErrchk(hipfftExecD2Z(*plan,
                                      static_cast<double*>(d_idata),
                                      static_cast<double2*>(d_odata)));
            break;
    }

    return;
}
