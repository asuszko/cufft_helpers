#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_r2c.h"


template<typename T>
inline hipfftResult_t cufftTR2C(hipfftHandle *plan,
                               const T *idata,
                               void *odata)
{
    if (std::is_same<T, float>::value) {
        return hipfftExecR2C(*plan,
                           (float *)idata,
                           (float2 *)odata);
    }
    else
    if (std::is_same<T, double>::value) {
        return hipfftExecD2Z(*plan,
                           (double *)idata,
                           (double2 *)odata);
    }
    else {
        return HIPFFT_EXEC_FAILED;
    }
}


/* C compatible version that requires a dtype_id to be converted
to the proper data type. */
void cufft_r2c(hipfftHandle *plan,
               const void *d_idata,
               void *d_odata,
               int dtype)
{

    switch(dtype) {

        case 2: {
            gpuFFTErrchk(cufftTR2C(plan,
                                  (float*)d_idata,
                                  d_odata));
            break;
        }

        case 3: {
            gpuFFTErrchk(cufftTR2C(plan,
                                  (double*)d_idata,
                                  d_odata));
            break;
        }
    }

    return;
}
