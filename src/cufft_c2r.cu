#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_c2r.h"


template<typename T>
inline hipfftResult_t cufftTC2R(hipfftHandle *plan,
                               const T *idata,
                               void *odata)
{
    if (std::is_same<T, float2>::value) {
        return hipfftExecC2R(*plan,
                           (float2 *)idata,
                           (float *)odata);
    }
    else
    if (std::is_same<T, double2>::value) {
        return hipfftExecZ2D(*plan,
                           (double2 *)idata,
                           (double *)odata);
    }
    else {
        return HIPFFT_EXEC_FAILED;
    }
}


/* C compatible version that requires a dtype_id to be converted
to the proper data type. */
void cufft_c2r(hipfftHandle *plan,
               const void *d_idata,
               void *d_odata,
               int dtype)
{
    switch(dtype) {

        case 2: {
            gpuFFTErrchk(cufftTC2R(plan,
                                  (float2 *)d_idata,
                                  d_odata));
            break;
        }

        case 3: {
            gpuFFTErrchk(cufftTC2R(plan,
                                  (double2 *)d_idata,
                                  d_odata));
            break;
        }
    }

    return;
}
