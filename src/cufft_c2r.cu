#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_c2r.h"



void cufft_c2r(hipfftHandle *plan,
               void *idata,
               void *odata,
               int dtype)
{
    switch(dtype) {

        case 2:
            gpuFFTErrchk(hipfftExecC2R(*plan,
                                      static_cast<float2*>(idata),
                                      static_cast<float*>(odata)));
            break;

        case 3:
            gpuFFTErrchk(hipfftExecZ2D(*plan,
                                      static_cast<double2*>(idata),
                                      static_cast<double*>(odata)));
            break;
    }

    return;
}
