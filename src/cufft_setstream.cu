#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_setstream.h"

void cufft_setstream(hipfftHandle *plan, hipStream_t *stream)
{
    if (stream == NULL) {
        hipfftSetStream(*plan, NULL);
    }
    else {
        hipfftSetStream(*plan, *stream);
    }
    return;
}
