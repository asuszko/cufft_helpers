#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_conj.h"


template<typename T>
__global__ void complex_conj(T *odata,
	                           int nx,
                             int ny,
                             int nz)
{
    int ix = threadIdx.x + blockDim.x * blockIdx.x;
    int iy = threadIdx.y + blockDim.y * blockIdx.y;
    int iz = threadIdx.z + blockDim.z * blockIdx.z;

    if (ix < nx && iy < ny && iz < nz) {
        odata[ix+iy*nx+iz*nx*ny].y *= -1.;
    }
}


void cufft_conj(void *d_data,
                dim3 extent,
                int dtype,
                hipStream_t *stream)
{
    int nx = extent.x;
    int ny = extent.y;
    int nz = extent.z;

    dim3 blockSize;
    if (nz > 1) {
        (nz <= 16) ? blockSize.z = nz : blockSize.z = 16;
    }
    if (ny > 1) {
        (ny <= 16) ? blockSize.y = ny : blockSize.y = 16;
    }
    if (nx > 1) {
        (nx <= 16) ? blockSize.x = ny : blockSize.x = 16;
    }

    while(blockSize.z*blockSize.y*blockSize.x > 1024) {
        (blockSize.y > blockSize.z) ? blockSize.y /= 2 : blockSize.z /= 2;
    }

    dim3 gridSize((nx-1)/blockSize.x+1,
                  (ny-1)/blockSize.y+1,
                  (nz-1)/blockSize.z+1);

    switch(dtype) {

        case 2: {
            complex_conj<<<gridSize,blockSize,0,*stream>>>((float2*)d_data,nx,ny,nz);
            break;
        }

        case 3: {
            complex_conj<<<gridSize,blockSize,0,*stream>>>((double2*)d_data,nx,ny,nz);
            break;
        }
    }

    return;
}
