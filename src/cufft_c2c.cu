#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_c2c.h"


#define CUFFTDIR(dir) ( (dir) == 0 ? HIPFFT_FORWARD : \
                        (dir) == 1 ? HIPFFT_BACKWARD : HIPFFT_BACKWARD)



void cufft_c2c(hipfftHandle *plan,
               void *idata,
               void *odata,
               int CUFFT_DIR,
               int dtype)
{
    switch(dtype) {

        case 2:
            gpuFFTErrchk(hipfftExecC2C(*plan,
                                      static_cast<float2*>(idata),
                                      static_cast<float2*>(odata),
                                      CUFFTDIR(CUFFT_DIR)));
            break;

        case 3:
            gpuFFTErrchk(hipfftExecZ2Z(*plan,
                                      static_cast<double2*>(idata),
                                      static_cast<double2*>(odata),
                                      CUFFTDIR(CUFFT_DIR)));
            break;
    }

    return;
}
