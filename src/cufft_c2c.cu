#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include "cu_errchk.h"
#include "cufft_c2c.h"


#define CUFFTDIR(dir) ( (dir) == 0 ? HIPFFT_FORWARD : \
                        (dir) == 1 ? HIPFFT_BACKWARD : HIPFFT_BACKWARD)


template<typename T>
inline hipfftResult_t cufftTC2C(hipfftHandle *plan,
                               const T *idata,
                               void *odata,
                               int fft_dir)
{
    if (std::is_same<T, float2>::value) {
        return hipfftExecC2C(*plan,
                           (float2 *)idata,
                           (float2 *)odata,
                           CUFFTDIR(fft_dir));
    }
    else
    if (std::is_same<T, double2>::value) {
        return hipfftExecZ2Z(*plan,
                           (double2 *)idata,
                           (double2 *)odata,
                           CUFFTDIR(fft_dir));
    }
    else {
        return HIPFFT_EXEC_FAILED;
    }
}



/* C compatible version that requires a dtype_id to be converted
to the proper data type. */
void cufft_c2c(hipfftHandle *plan,
               const void *d_idata,
               void *d_odata,
               int CUFFT_DIR,
               int dtype)
{
    switch(dtype) {

          case 2: {
              gpuFFTErrchk(cufftTC2C(plan,
                                    (float2*)d_idata,
                                    d_odata,
                                    CUFFT_DIR));
              break;
          }

          case 3: {
              gpuFFTErrchk(cufftTC2C(plan,
                                    (double2*)d_idata,
                                    d_odata,
                                    CUFFT_DIR));
              break;
          }
      }

      return;
}
